#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "cnn_naive.cuh"  // Include the header file with the naive kernel

// Function to launch the convolution kernel
void cnn_naive(float *h_input, float *h_output, float *h_mask,
               int dimX, int dimY, int dimK) {
    float *d_input, *d_output, *d_mask;
    size_t img_size = dimX * dimY * sizeof(float);
    size_t mask_size = dimK * dimK * sizeof(float);
    
    // Allocate device memory
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMalloc((void**)&d_mask, mask_size);
    
    // Copy data to device
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice);
    
    // Set grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, 
                 (dimY + blockDim.y - 1) / blockDim.y);
    
    // Use the performance measurement function
    float gflops = measurePerformance(d_input, d_mask, d_output, dimX, dimY, dimK, gridDim, blockDim);

    // Copy result back to host
    hipMemcpy(h_output, d_output, img_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
