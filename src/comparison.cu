#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>

#include "../include/utils.cuh"
#include "../include/histogram_naive.cuh"        // Kernel: histogram_naive_kernel(const int*, int*, int, int)
#include "../include/histogram_optimized.cuh"    // Kernel: histogram_optimized_kernel(const int*, int*, int, int)
#include "../include/histogram_tiling.cuh"         // Kernel: histogram_tiled_kernel(const int*, int*, int, int)
// Reduction kernel is assumed common for optimized and tiled kernels.
extern __global__ void histogram_reduce_kernel(const int *partialHist, int *finalHist, int numBins, int numBlocks);

using namespace cooperative_groups;

int main(int argc, char *argv[]) {
    // Usage: ./histogram_atomic -i <BinNum> <VecDim> [GridSize]
    if (argc < 4 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        fprintf(stderr, "Usage: %s -i <BinNum> <VecDim> [GridSize]\n", argv[0]);
        return 1;
    }
    
    int numBins = atoi(argv[2]);
    int N = atoi(argv[3]);
    
    // Validate numBins: must be 2^k with k between 2 and 8.
    if (numBins < 4 || numBins > 256 || (numBins & (numBins - 1)) != 0) {
        fprintf(stderr, "Error: <BinNum> must be 2^k with k from 2 to 8 (e.g., 4, 8, 16, 32, 64, 128, or 256).\n");
        return 1;
    }
    
    // With fixed block dimensions (e.g., 32x32 = 1024 threads), we set block and grid.
    const int blockSizeTotal = 8 * 32;  // For example, 256 threads per block.
    int gridSize;
    if (argc >= 5)
        gridSize = atoi(argv[4]);
    else
        gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;
    
    // Set block and grid dimensions.
    dim3 block(4, 64);  // 4*64=256 threads per block.
    dim3 grid(gridSize);
    
    // Compute shared memory sizes.
    // For optimized kernel:
    //   Two tile buffers: tileSizeInts = block.x * block.y * 4 integers.
    //   Plus per-warp histogram: numWarps * numBins integers.
    int tileSizeInts = block.x * block.y * 4;
    int numWarps = (block.x * block.y) / 32;
    size_t sharedMemSize_optimized = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
    // For tiled kernel, we only need per-warp histogram shared memory.
    size_t sharedMemSize_tiled = (numWarps * numBins) * sizeof(int);
    // For naive kernel, shared memory is for one histogram per block.
    size_t sharedMemSize_naive = numBins * sizeof(int);
    
    // Data sizes.
    size_t dataSize = N * sizeof(int);
    size_t partialHistSize = gridSize * numBins * sizeof(int);
    size_t finalHistSize = numBins * sizeof(int);
    
    // Allocate and initialize host memory.
    int *h_data = (int*) malloc(dataSize);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory for input data.\n");
        return 1;
    }
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_data[i] = rand() % 1024;  // values in [0, 1023]
    }
    
    // Allocate device memory.
    int *d_data, *d_partialHist, *d_finalHist;
    hipMalloc((void**)&d_data, dataSize);
    hipMalloc((void**)&d_partialHist, partialHistSize);
    hipMalloc((void**)&d_finalHist, finalHistSize);
    
    // Copy input data to device.
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    
    // Create CUDA events.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // --- 1) Naive Kernel ---
    hipMemset(d_finalHist, 0, finalHistSize);
    hipEventRecord(start, 0);
    
    // Launch naive kernel. (Naive kernel writes directly to finalHist.)
    histogram_naive_kernel<<<grid, block, sharedMemSize_naive>>>(d_data, d_finalHist, N, numBins);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeNaive;
    hipEventElapsedTime(&elapsedTimeNaive, start, stop);
    
    // Calculate measured throughput based on approximate atomic operations.
    // Per-element atomic operations: ~3 per element.
    // Final merge is done inside kernel (for naive) so no extra reduction.
    double totalOps = 3.0 * N + ((double)N / tileSizeInts) * numBins;
    double elapsedSecNaive = elapsedTimeNaive / 1000.0;
    double opsPerSecNaive = totalOps / elapsedSecNaive;
    double measuredGopsNaive = opsPerSecNaive / 1e9;
    
    printf("\n=== Naive Kernel ===\n");
    printf("Total execution time: %f ms\n", elapsedTimeNaive);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSecNaive);
    printf("Measured Performance: %f Gops (atomic ops metric)\n", measuredGopsNaive);
    
    
    // --- 2) Optimized Kernel ---
    // Reset device buffers.
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);
    hipEventRecord(start, 0);
    
    // Launch optimized kernel (double buffering etc.)
    histogram_optimized_kernel<<<grid, block, sharedMemSize_optimized>>>(d_data, d_partialHist, N, numBins);
    // Launch reduction kernel to sum partial histograms.
    int reduceBlockSize = 256;
    int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
    histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeOptimized;
    hipEventElapsedTime(&elapsedTimeOptimized, start, stop);
    
    double elapsedSecOptimized = elapsedTimeOptimized / 1000.0;
    double opsPerSecOptimized = totalOps / elapsedSecOptimized;
    double measuredGopsOptimized = opsPerSecOptimized / 1e9;
    
    printf("\n=== Optimized Kernel ===\n");
    printf("Total execution time: %f ms\n", elapsedTimeOptimized);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSecOptimized);
    printf("Measured Performance: %f Gops (atomic ops metric)\n", measuredGopsOptimized);
    
    
    // --- 3) Tiled Kernel ---
    // Reset device buffers.
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);
    hipEventRecord(start, 0);
    
    // Launch tiled kernel.
    histogram_tiled_kernel<<<grid, block, sharedMemSize_tiled>>>(d_data, d_partialHist, N, numBins);
    // Launch reduction kernel.
    histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeTiled;
    hipEventElapsedTime(&elapsedTimeTiled, start, stop);
    
    double elapsedSecTiled = elapsedTimeTiled / 1000.0;
    double opsPerSecTiled = totalOps / elapsedSecTiled;
    double measuredGopsTiled = opsPerSecTiled / 1e9;
    
    printf("\n=== Tiled Kernel ===\n");
    printf("Total execution time: %f ms\n", elapsedTimeTiled);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSecTiled);
    printf("Measured Performance: %f Gops (atomic ops metric)\n", measuredGopsTiled);
    
    
    // Calculate occupancy for the optimized kernel (as an example).
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerSM = deviceProp.maxThreadsPerMultiProcessor;
    int activeBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeBlocks, histogram_optimized_kernel, blockSizeTotal, sharedMemSize_optimized);
    float occupancy = (activeBlocks * blockSizeTotal) / (float) maxThreadsPerSM;
    occupancy = occupancy * 100.0f;
    printf("\nOccupancy per SM (Optimized Kernel): %f %%\n", occupancy);
     
    // Display device properties.
    int coresPerSM = 64;  // adjust if needed per architecture.
    int totalCores = deviceProp.multiProcessorCount * coresPerSM;
    double clockHz = deviceProp.clockRate * 1000.0;
    double theoreticalOps = totalCores * clockHz * 2;
    printf("Device: %s\n", deviceProp.name);
    printf("Number of SMs: %d\n", deviceProp.multiProcessorCount);
    printf("Cores per SM: %d\n", coresPerSM);
    printf("Total CUDA Cores: %d\n", totalCores);
    printf("Clock Rate: %0.2f GHz\n", clockHz / 1e9);
    printf("Theoretical Peak Ops/sec (int): %e ops/sec\n", theoreticalOps);
    
    // (Optional) Copy final histogram from device to host and print nonzero bins.
    int *h_finalHist = (int*) malloc(finalHistSize);
    hipMemcpy(h_finalHist, d_finalHist, finalHistSize, hipMemcpyDeviceToHost);
    printf("\nFinal Histogram (nonzero bins):\n");
    for (int i = 0; i < numBins; i++) {
        if (h_finalHist[i] != 0)
            printf("Bin %d: %d\n", i, h_finalHist[i]);
    }
    
    // Clean up.
    free(h_data);
    free(h_finalHist);
    hipFree(d_data);
    hipFree(d_partialHist);
    hipFree(d_finalHist);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}