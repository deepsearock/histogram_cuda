#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>

#include "../include/utils.cuh"
#include "../include/histogram_naive.cuh"        // Kernel: histogram_naive_kernel(const int*, int*, int, int)
#include "../include/histogram_optimized.cuh"    // Kernel: histogram_optimized_kernel(const int*, int*, int, int)
#include "../include/histogram_tiling.cuh"         // Kernel: histogram_tiled_kernel(const int*, int*, int, int)
// Reduction kernel used for optimized and tiled kernels.
extern __global__ void histogram_reduce_kernel(const int *partialHist, int *finalHist, int numBins, int numBlocks);

using namespace cooperative_groups;

int main(int argc, char *argv[]) {
    // Usage: ./graph -i <VecDim> [GridSize]
    if (argc < 3 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        fprintf(stderr, "Usage: %s -i <VecDim> [GridSize]\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[2]);    // vector dimension / number of elements
    int gridSize;
    const int blockSizeTotal = 8 * 32; // 256 threads per block
    if (argc >= 4)
        gridSize = atoi(argv[3]);
    else
        gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;
    
    // Set block and grid dimensions.
    dim3 block(4, 64);   // 4 x 64 = 256 threads per block.
    dim3 grid(gridSize);
    
    // Shared memory sizes for each kernel:
    // Naive kernel: one histogram per block.
    size_t sharedMemSize_naive = 0; // no extra shared mem allocated, as the kernel allocates histogram of 'numBins' internally.
    // For optimized kernel:
    int tileSizeInts = block.x * block.y * 4;
    int numWarps = (block.x * block.y) / 32;
    size_t sharedMemSize_optimized = (2 * tileSizeInts + numWarps * /*numBins*/256) * sizeof(int);
    // For tiled kernel: only per-warp histogram.
    size_t sharedMemSize_tiled = (numWarps * /*numBins*/256) * sizeof(int);
    // (Note: These shared mem sizes will be adjusted inside the loop below based on numBins.)
    
    // Data sizes.
    size_t dataSize = N * sizeof(int);
    
    // Allocate and initialize host data.
    int *h_data = (int*) malloc(dataSize);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory for input data.\n");
        return 1;
    }
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_data[i] = rand() % 1024; // values in [0, 1023]
    }
    
    // Allocate device memory.
    int *d_data, *d_partialHist, *d_finalHist;
    hipMalloc((void**)&d_data, dataSize);
    // We'll allocate d_partialHist and d_finalHist based on maximum possible sizes.
    // Maximum partialHist is gridSize * max(numBins) integers.
    int maxBins = 256;
    size_t partialHistSize = gridSize * maxBins * sizeof(int);
    size_t finalHistSize = maxBins * sizeof(int);
    hipMalloc((void**)&d_partialHist, partialHistSize);
    hipMalloc((void**)&d_finalHist, finalHistSize);
    
    // Copy input data to device.
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    
    // Open CSV file for writing results.
    FILE *fp = fopen("results.csv", "w");
    if (!fp) {
        fprintf(stderr, "Failed to open CSV file for writing.\n");
        return 1;
    }
    // Write CSV header.
    fprintf(fp, "Kernel,NumBins,ExecutionTime_ms,TotalOps,Throughput_ops_sec,Gops\n");
    
    // Define the bin sizes to test.
    const int binSizes[7] = {4, 8, 16, 32, 64, 128, 256};
    // Calculate a common total operations estimate.
    double totalOps = 3.0 * N + ((double)N / (block.x * block.y * 4)) * maxBins; // using maxBins for estimation
    
    // Create CUDA events.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // For each kernel variant, loop through each bin setting.
    
    // --- Naive Kernel ---
    for (int bi = 0; bi < 7; bi++) {
        int numBins = binSizes[bi];
        // Reset final histogram.
        hipMemset(d_finalHist, 0, numBins * sizeof(int));
        
        hipEventRecord(start, 0);
        // Launch naive kernel.
        // Note: histogram_naive_kernel writes directly to finalHist,
        // so we pass numBins and use sharedMemSize_naive as configured in the kernel.
        histogram_naive_kernel<<<grid, block, numBins * sizeof(int)>>>(d_data, d_finalHist, N, numBins);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        double elapsedSec = elapsedTime / 1000.0;
        double opsPerSec = totalOps / elapsedSec;
        double measuredGops = opsPerSec / 1e9;
        
        fprintf(fp, "Naive,%d,%f,%.0f,%e,%f\n", numBins, elapsedTime, totalOps, opsPerSec, measuredGops);
    }
    
    // --- Optimized Kernel ---
    for (int bi = 0; bi < 7; bi++) {
        int numBins = binSizes[bi];
        // Adjust shared memory size for optimized kernel.
        size_t sharedMemSizeOpt = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
        // Reset device buffers.
        hipMemset(d_partialHist, 0, gridSize * numBins * sizeof(int));
        hipMemset(d_finalHist, 0, numBins * sizeof(int));
        
        hipEventRecord(start, 0);
        // Launch optimized kernel.
        histogram_optimized_kernel<<<grid, block, sharedMemSizeOpt>>>(d_data, d_partialHist, N, numBins);
        // Launch reduction kernel.
        int reduceBlockSize = 256;
        int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
        histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        double elapsedSec = elapsedTime / 1000.0;
        double opsPerSec = totalOps / elapsedSec;
        double measuredGops = opsPerSec / 1e9;
        
        fprintf(fp, "Optimized,%d,%f,%.0f,%e,%f\n", numBins, elapsedTime, totalOps, opsPerSec, measuredGops);
    }
    
    // --- Tiled Kernel ---
    for (int bi = 0; bi < 7; bi++) {
        int numBins = binSizes[bi];
        // Adjust shared memory size for tiled kernel.
        size_t sharedMemSizeTile = (numWarps * numBins) * sizeof(int);
        // Reset device buffers.
        hipMemset(d_partialHist, 0, gridSize * numBins * sizeof(int));
        hipMemset(d_finalHist, 0, numBins * sizeof(int));
        
        hipEventRecord(start, 0);
        // Launch tiled kernel.
        histogram_tiled_kernel<<<grid, block, sharedMemSizeTile>>>(d_data, d_partialHist, N, numBins);
        // Launch reduction kernel.
        int reduceBlockSize = 256;
        int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
        histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        double elapsedSec = elapsedTime / 1000.0;
        double opsPerSec = totalOps / elapsedSec;
        double measuredGops = opsPerSec / 1e9;
        
        fprintf(fp, "Tiled,%d,%f,%.0f,%e,%f\n", numBins, elapsedTime, totalOps, opsPerSec, measuredGops);
    }
    
    fclose(fp);
    
    // Clean up.
    free(h_data);
    hipFree(d_data);
    hipFree(d_partialHist);
    hipFree(d_finalHist);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}