#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>

#include "../include/histogram_optimized.cuh"
#include "../include/utils.cuh"
namespace cg = cooperative_groups;

int main(int argc, char *argv[]) {
    // input
    if (argc < 4 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        fprintf(stderr, "Usage: %s -i <BinNum> <VecDim> [GridSize]\n", argv[0]);
        return 1;
    }
    
    int numBins = atoi(argv[2]);
    int N = atoi(argv[3]);
    
    // handle error
    if (numBins < 4 || numBins > 256 || (numBins & (numBins - 1)) != 0) {
        fprintf(stderr, "Error: <BinNum> must be 2^k with k from 2 to 8 (e.g., 4, 8, 16, 32, 64, 128, or 256).\n");
        return 1;
    }


    // block and grid size
    const int blockSizeTotal = 8 * 32;
    int gridSize;
    if (argc >= 5)
        gridSize = atoi(argv[4]);
    else
        gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;

    dim3 block(4, 64);
    dim3 grid(gridSize);
    

    int tileSizeInts = block.x * block.y * 4;
    int numWarps = (block.x * block.y) / 32;
    size_t sharedMemSize = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
    
    size_t dataSize = N * sizeof(int);
    size_t partialHistSize = gridSize * numBins * sizeof(int);
    size_t finalHistSize = numBins * sizeof(int);

    //calculate shared memory size then return error if too big
    int *h_data = (int*) malloc(dataSize);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory for input data.\n");
        return 1;
    }
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_data[i] = rand() % 1024;
    }


    //allocate memory
    int *d_data, *d_partialHist, *d_finalHist;
    hipMalloc((void**)&d_data, dataSize);
    hipMalloc((void**)&d_partialHist, partialHistSize);
    hipMalloc((void**)&d_finalHist, finalHistSize);

    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);


    //start kernel measure performance 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);

    histogram_optimized_kernel<<<grid, block, sharedMemSize>>>(d_data, d_partialHist, N, numBins);

    int reduceBlockSize = 256;
    int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
    histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    double totalOps = 3.0 * N + ((double)N / tileSizeInts) * numBins;
    double elapsedSec = elapsedTime / 1000.0;
    double opsPerSec = totalOps / elapsedSec;
    double measuredGops = opsPerSec / 1e9;
    
    printf("Total kernel execution time: %f ms\n", elapsedTime);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSec);
    printf("Measured Performance: %f Gops (atomic ops metric)\n", measuredGops);


    // get device data
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerSM = deviceProp.maxThreadsPerMultiProcessor;
    int activeBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeBlocks, histogram_optimized_kernel, blockSizeTotal, sharedMemSize);
    float occupancy = (activeBlocks * blockSizeTotal) / (float) maxThreadsPerSM;
    occupancy = occupancy * 100.0f;
    printf("Occupancy per SM: %f %%\n", occupancy);

    int coresPerSM = 64;
    int totalCores = deviceProp.multiProcessorCount * coresPerSM;
    double clockHz = deviceProp.clockRate * 1000.0;
    double theoreticalOps = totalCores * clockHz * 2;
    printf("Device: %s\n", deviceProp.name);
    printf("Number of SMs: %d\n", deviceProp.multiProcessorCount);
    printf("Cores per SM: %d\n", coresPerSM);
    printf("Total CUDA Cores: %d\n", totalCores);
    printf("Clock Rate: %0.2f GHz\n", clockHz / 1e9);
    printf("Theoretical Peak Ops/sec (int): %e ops/sec\n", theoreticalOps);


    // check kernel
    int *h_finalHist = (int*) malloc(finalHistSize);
    hipMemcpy(h_finalHist, d_finalHist, finalHistSize, hipMemcpyDeviceToHost);
    for (int i = 0; i < numBins; i++) {
        if (h_finalHist[i] != 0)
            printf("Bin %d: %d\n", i, h_finalHist[i]);
    }

    free(h_data);
    free(h_finalHist);
    hipFree(d_data);
    hipFree(d_partialHist);
    hipFree(d_finalHist);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}