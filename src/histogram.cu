#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

// Define a prefetch function using inline PTX for global memory.
__device__ inline void prefetch_global(const void *ptr) {
    asm volatile("prefetch.global.L1 [%0];" :: "l"(ptr));
}

// Optimized histogram kernel with double buffering, vectorized loads,
// per-warp histograms, bit-shift based bin calculation, and loop unrolling.
__global__ void histogram_optimized_kernel(const int *data, int *partialHist, int N, int numBins) {
    // Shared memory layout:
    // [tile0 | tile1 | per-warp histograms]
    extern __shared__ int sharedMem[];

    const int warpSize = 32;
    int blockThreads = blockDim.x * blockDim.y;
    // Each tile holds blockThreads * 4 integers (vectorized loads: int4)
    int tileSizeInts = blockThreads * 4;
    int *tile0 = sharedMem;                      // first tile buffer
    int *tile1 = sharedMem + tileSizeInts;         // second tile buffer
    int numWarps = blockThreads / warpSize;        // assume blockThreads is a multiple of 32
    int *warpHist = (int*)(sharedMem + 2 * tileSizeInts); // per-warp histogram region

    // Precompute the bit-shift factor.
    // Since 1024 is the max value and numBins is 2^k, each bin spans 1024/numBins values.
    // log2(1024/numBins) = 10 - log2(numBins)
    int k = 0;
    int temp = numBins;
    while (temp > 1) {
        k++;
        temp >>= 1;
    }
    int shift = 10 - k;  // e.g., if numBins = 8 (k=3), then shift = 7.

    // Flatten the thread index.
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int warp_id = tid / warpSize;
    int lane = tid % warpSize;

    // Initialize per-warp histograms.
    for (int i = lane; i < numWarps * numBins; i += warpSize) {
        warpHist[i] = 0;
    }
    __syncthreads();

    // Compute global tile size (in ints) for double buffering.
    int globalTileSizeInts = gridDim.x * tileSizeInts;
    int firstOffset = blockIdx.x * tileSizeInts;

    // Load the first tile from global memory into tile0 using __ldg and prefetching.
    if (firstOffset < N) {
        int globalIndex = firstOffset + tid * 4;
        // Prefetch future data (64 is an arbitrary offset; adjust as needed)
        prefetch_global(&data[globalIndex + 64]);
        if (globalIndex + 3 < N) {
            int4 tmp = __ldg(reinterpret_cast<const int4*>(&data[globalIndex]));
            tile0[tid * 4 + 0] = tmp.x;
            tile0[tid * 4 + 1] = tmp.y;
            tile0[tid * 4 + 2] = tmp.z;
            tile0[tid * 4 + 3] = tmp.w;
        } else {
            for (int i = 0; i < 4; i++) {
                int idx = globalIndex + i;
                tile0[tid * 4 + i] = (idx < N) ? __ldg(&data[idx]) : -1;
            }
        }
    }
    __syncthreads();

    // Process the tiles using double buffering.
    // In the double-buffering loop, prefetch the next tile and load using __ldg.
    for (int offset = firstOffset + globalTileSizeInts; offset < N; offset += globalTileSizeInts) {
        int globalIndex = offset + tid * 4;
        // Prefetch a future block of data. Adjust "64" as a prefetch distance.
        prefetch_global(&data[globalIndex + 64]);
        if (globalIndex + 3 < N) {
            int4 tmp = __ldg(reinterpret_cast<const int4*>(&data[globalIndex]));
            tile1[tid * 4 + 0] = tmp.x;
            tile1[tid * 4 + 1] = tmp.y;
            tile1[tid * 4 + 2] = tmp.z;
            tile1[tid * 4 + 3] = tmp.w;
        } else {
            for (int i = 0; i < 4; i++) {
                int idx = globalIndex + i;
                tile1[tid * 4 + i] = (idx < N) ? __ldg(&data[idx]) : -1;
            }
        }
        __syncthreads();

        // Process the current tile (in tile0) using a per-thread run-length aggregation.
        {
            int localBin = -1;
            int localCount = 0;
            #pragma unroll
            for (int i = tid; i < tileSizeInts; i += blockThreads) {
                int value = tile0[i];
                if (value < 0) continue;
                // Use bit shift instead of division.
                int bin = value >> shift;
                if (bin == localBin) {
                    localCount++;
                } else {
                    if (localCount > 0)
                        atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
                    localBin = bin;
                    localCount = 1;
                }
            }
            if (localCount > 0)
                atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
        }
        __syncthreads();

        // Swap tile buffers.
        int *tempPtr = tile0;
        tile0 = tile1;
        tile1 = tempPtr;
        __syncthreads();
    }

    // Process the final tile loaded in tile0.
    {
        // Assume numBins is small enough; here we allocate a fixed-size array for local counts.
        // For maximum flexibility, we allocate up to 256 entries (since k ∈ [2,8]).
        int localHist[256];
        #pragma unroll
        for (int b = 0; b < numBins; b++) {
            localHist[b] = 0;
        }

        // Each thread processes multiple elements from the current tile.
        for (int i = tid; i < tileSizeInts; i += blockThreads) {
            int value = tile0[i];
            if (value < 0) continue;
            // Compute bin index via bit-shift.
            int bin = value >> shift;
            if(bin < numBins)
                localHist[bin]++;
        }

        // For each bin, perform a warp-level reduction using shuffles so that only one thread per warp
        // issues an atomicAdd to the per-warp histogram.
        unsigned mask = 0xffffffff;  // Full warp
        for (int bin = 0; bin < numBins; bin++) {
            int sum = localHist[bin];
            // Reduce across the warp.
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                sum += __shfl_down_sync(mask, sum, offset);
            }
            // Only lane 0 in each warp contributes to the global per-warp histogram.
            if(lane == 0) {
                atomicAdd(&warpHist[warp_id * numBins + bin], sum);
            }
        }
    }
    __syncthreads();

    // Reduce the per-warp histograms into a block-level (partial) histogram.
    if (warp_id == 0) {
        for (int i = lane; i < numBins; i += warpSize) {
            int sum = 0;
            for (int w = 0; w < numWarps; w++) {
                sum += warpHist[w * numBins + i];
            }
            partialHist[blockIdx.x * numBins + i] = sum;
        }
    }
}

// Reduction kernel: Sum partial histograms from all blocks into the final histogram.
__global__ void histogram_reduce_kernel(const int *partialHist, int *finalHist, int numBins, int numBlocks) {
    int bin = blockIdx.x * blockDim.x + threadIdx.x;
    if (bin < numBins) {
        int sum = 0;
        for (int b = 0; b < numBlocks; b++) {
            sum += partialHist[b * numBins + bin];
        }
        finalHist[bin] = sum;
    }
}

int main(int argc, char *argv[]) {
    // Usage: ./histogram_atomic -i <BinNum> <VecDim> [GridSize]
    if (argc < 4 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        fprintf(stderr, "Usage: %s -i <BinNum> <VecDim> [GridSize]\n", argv[0]);
        return 1;
    }
    
    int numBins = atoi(argv[2]);
    int N = atoi(argv[3]);
    
    // Validate numBins: must be 2^k with k between 2 and 8.
    if (numBins < 4 || numBins > 256 || (numBins & (numBins - 1)) != 0) {
        fprintf(stderr, "Error: <BinNum> must be 2^k with k from 2 to 8 (e.g., 4, 8, 16, 32, 64, 128, or 256).\n");
        return 1;
    }
    
    // With fixed block dimensions (32x32), total threads per block is 1024.
    const int blockSizeTotal = 8 * 32;
    int gridSize;
    if (argc >= 5)
        gridSize = atoi(argv[4]);
    else
        gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;
    
    // Set fixed block dimensions: 32 x 32.
    dim3 block(4, 64);
    dim3 grid(gridSize);
    
    // Calculate shared memory size:
    // Two tile buffers: 2 * tileSizeInts, where tileSizeInts = block.x * block.y * 4.
    // Plus per-warp histogram: numWarps * numBins integers.
    int tileSizeInts = block.x * block.y * 4;
    int numWarps = (block.x * block.y) / 32;
    size_t sharedMemSize = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
    
    size_t dataSize = N * sizeof(int);
    size_t partialHistSize = gridSize * numBins * sizeof(int);
    size_t finalHistSize = numBins * sizeof(int);
    
    // Allocate and initialize host memory.
    int *h_data = (int*) malloc(dataSize);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory for input data.\n");
        return 1;
    }
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_data[i] = rand() % 1024;  // values in [0, 1023]
    }
    
    // Allocate device memory.
    int *d_data, *d_partialHist, *d_finalHist;
    hipMalloc((void**)&d_data, dataSize);
    hipMalloc((void**)&d_partialHist, partialHistSize);
    hipMalloc((void**)&d_finalHist, finalHistSize);
    
    // Copy input data to device.
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);
    
    // Create CUDA events.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    
    // Launch the optimized histogram kernel.
    histogram_optimized_kernel<<<grid, block, sharedMemSize>>>(d_data, d_partialHist, N, numBins);
    
    // Launch the reduction kernel.
    int reduceBlockSize = 256;
    int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
    histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    // Calculate measured throughput based on approximate atomic operations.
    // Per-element operations: ~3 operations per element
    // Final merge: gridSize * numBins operations.
    double totalOps = 3.0 * N + (gridSize * numBins);
    double elapsedSec = elapsedTime / 1000.0;
    double opsPerSec = totalOps / elapsedSec;
    double measuredGFlops = opsPerSec / 1e9;
    
    printf("Total kernel execution time: %f ms\n", elapsedTime);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSec);
    printf("Measured Performance: %f GFLOPS (atomic ops metric)\n", measuredGFlops);
    
    // Calculate occupancy.
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerSM = deviceProp.maxThreadsPerMultiProcessor;
    int activeBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeBlocks, histogram_optimized_kernel, blockSizeTotal, sharedMemSize);
    float occupancy = (activeBlocks * blockSizeTotal) / (float) maxThreadsPerSM;
    occupancy = occupancy * 100.0f;
    printf("Occupancy per SM: %f %%\n", occupancy);
    
    // Display device properties.
    int coresPerSM = 64;
    int totalCores = deviceProp.multiProcessorCount * coresPerSM;
    double clockHz = deviceProp.clockRate * 1000.0;
    double theoreticalOps = totalCores * clockHz * 2;
    printf("Device: %s\n", deviceProp.name);
    printf("Number of SMs: %d\n", deviceProp.multiProcessorCount);
    printf("Cores per SM: %d\n", coresPerSM);
    printf("Total CUDA Cores: %d\n", totalCores);
    printf("Clock Rate: %0.2f GHz\n", clockHz / 1e9);
    printf("Theoretical Peak Ops/sec (int): %e ops/sec\n", theoreticalOps);
    
    // (Optional) Copy final histogram from device to host and print nonzero bins.
    int *h_finalHist = (int*) malloc(finalHistSize);
    hipMemcpy(h_finalHist, d_finalHist, finalHistSize, hipMemcpyDeviceToHost);
    for (int i = 0; i < numBins; i++) {
        if (h_finalHist[i] != 0)
            printf("Bin %d: %d\n", i, h_finalHist[i]);
    }
    
    // Clean up.
    free(h_data);
    free(h_finalHist);
    hipFree(d_data);
    hipFree(d_partialHist);
    hipFree(d_finalHist);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
