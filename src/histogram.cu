#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

// Optimized histogram kernel with double buffering, vectorized loads,
// per-warp histograms, bit-shift based bin calculation, and loop unrolling.
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

__global__ void histogram_optimized_kernel(const int * __restrict__ data, int *partialHist, int N, int numBins) {
    // Shared memory layout:
    // [tile0 | tile1 | per-warp histograms]
    extern __shared__ int sharedMem[];

    const int warpSize = 32;
    int blockThreads = blockDim.x * blockDim.y;
    // Each tile holds blockThreads * 4 integers (using int4 loads).
    int tileSizeInts = blockThreads * 4;
    int *tile0 = sharedMem;                      // first tile buffer
    int *tile1 = sharedMem + tileSizeInts;         // second tile buffer
    int numWarps = blockThreads / warpSize;        // assume blockThreads is a multiple of 32
    int *warpHist = sharedMem + 2 * tileSizeInts;    // per-warp histogram region

    // Precompute the bit-shift factor.
    int k = 0;
    int tmp = numBins;
    while (tmp > 1) {
        k++;
        tmp >>= 1;
    }
    int shift = 10 - k;  // e.g., if numBins = 8 (k=3), then shift = 7.

    // Flatten the thread index.
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int warp_id = tid / warpSize;
    int lane = tid % warpSize;

    // Initialize per-warp histograms.
    for (int i = lane; i < numWarps * numBins; i += warpSize) {
        warpHist[i] = 0;
    }
    __syncthreads();

    // Compute global tile size (in ints) for double buffering.
    int globalTileSizeInts = gridDim.x * tileSizeInts;
    int firstOffset = blockIdx.x * tileSizeInts;

    // --- Load the first tile into tile0 synchronously using vectorized loads ---
    if (firstOffset < N) {
        int globalIndex = firstOffset + tid * 4;
        if (globalIndex + 3 < N) {
            // Use __ldg to force use of the read-only cache.
            int4 tmpVal = ((const int4 *)__ldg(&data[globalIndex/4]))[0];
            tile0[tid * 4 + 0] = tmpVal.x;
            tile0[tid * 4 + 1] = tmpVal.y;
            tile0[tid * 4 + 2] = tmpVal.z;
            tile0[tid * 4 + 3] = tmpVal.w;
        } else {
            for (int i = 0; i < 4; i++) {
                int idx = globalIndex + i;
                tile0[tid * 4 + i] = (idx < N) ? data[idx] : -1;
            }
        }
    }
    __syncthreads();

    // --- Process tiles with double buffering using asynchronous copy ---
    // On Volta we can use cp.async to prefetch the next tile into tile1.
    for (int offset = firstOffset + globalTileSizeInts; offset < N; offset += globalTileSizeInts) {
        int globalIndex = offset + tid * 4;
        // Asynchronously copy one int4 (16 bytes) from global memory to shared memory.
        if (globalIndex + 3 < N) {
            // cp.async.cg.shared.global copies 16 bytes.
            // The destination is tile1 + (tid * 4) (in bytes) and source is data + globalIndex.
            asm volatile (
                "cp.async.cg.shared.global [%0], [%1], %2;\n"
                :: "r"(tile1 + tid*4),
                   "l"(data + globalIndex),
                   "n"(16)
            );
        } else {
            // Fallback: perform scalar loads.
            for (int i = 0; i < 4; i++) {
                int idx = globalIndex + i;
                tile1[tid * 4 + i] = (idx < N) ? data[idx] : -1;
            }
        }
        // Commit the asynchronous copy group.
        asm volatile ("cp.async.commit_group;\n");
        __syncthreads();
        // Wait for all async copies in the group to complete.
        asm volatile ("cp.async.wait_group 0;\n");
        __syncthreads();

        // Process the current tile (in tile0) using a per-thread run-length aggregation.
        {
            int localBin = -1;
            int localCount = 0;
            #pragma unroll
            for (int i = tid; i < tileSizeInts; i += blockThreads) {
                int value = tile0[i];
                if (value < 0) continue;
                int bin = value >> shift;
                if (bin == localBin) {
                    localCount++;
                } else {
                    if (localCount > 0)
                        atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
                    localBin = bin;
                    localCount = 1;
                }
            }
            if (localCount > 0)
                atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
        }
        __syncthreads();

        // Swap tile buffers so that tile1 (newly loaded) becomes the current tile.
        int *tempPtr = tile0;
        tile0 = tile1;
        tile1 = tempPtr;
        __syncthreads();
    }

    // --- Process the final tile loaded in tile0 ---
    {
        int localBin = -1;
        int localCount = 0;
        #pragma unroll
        for (int i = tid; i < tileSizeInts; i += blockThreads) {
            int value = tile0[i];
            if (value < 0) continue;
            int bin = value >> shift;
            if (bin == localBin) {
                localCount++;
            } else {
                if (localCount > 0)
                    atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
                localBin = bin;
                localCount = 1;
            }
        }
        if (localCount > 0)
            atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
    }
    __syncthreads();

    // Reduce the per-warp histograms into a block-level (partial) histogram.
    if (warp_id == 0) {
        for (int i = lane; i < numBins; i += warpSize) {
            int sum = 0;
            for (int w = 0; w < numWarps; w++) {
                sum += warpHist[w * numBins + i];
            }
            partialHist[blockIdx.x * numBins + i] = sum;
        }
    }
}


// Reduction kernel: Sum partial histograms from all blocks into the final histogram.
__global__ void histogram_reduce_kernel(const int *partialHist, int *finalHist, int numBins, int numBlocks) {
    int bin = blockIdx.x * blockDim.x + threadIdx.x;
    if (bin < numBins) {
        int sum = 0;
        for (int b = 0; b < numBlocks; b++) {
            sum += partialHist[b * numBins + bin];
        }
        finalHist[bin] = sum;
    }
}

int main(int argc, char *argv[]) {
    // Usage: ./histogram_atomic -i <BinNum> <VecDim> [GridSize]
    // Note: With a fixed block dimension of 32x32, total threads per block is 1024.
    if (argc < 4 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        fprintf(stderr, "Usage: %s -i <BinNum> <VecDim> [GridSize]\n", argv[0]);
        return 1;
    }
    
    int numBins = atoi(argv[2]);
    int N = atoi(argv[3]);
    
    // Validate numBins: must be 2^k with k between 2 and 8.
    if (numBins < 4 || numBins > 256 || (numBins & (numBins - 1)) != 0) {
        fprintf(stderr, "Error: <BinNum> must be 2^k with k from 2 to 8 (e.g., 4, 8, 16, 32, 64, 128, or 256).\n");
        return 1;
    }
    
    // With fixed block dimensions (32x32), total threads per block is 1024.
    const int blockSizeTotal = 8 * 32;
    int gridSize;
    if (argc >= 5)
        gridSize = atoi(argv[4]);
    else
        gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;
    
    // Set fixed block dimensions: 32 x 32.
    dim3 block(8, );
    dim3 grid(gridSize);
    
    // Calculate shared memory size:
    // Two tile buffers: 2 * tileSizeInts, where tileSizeInts = block.x * block.y * 4.
    // Plus per-warp histogram: numWarps * numBins integers.
    int tileSizeInts = block.x * block.y * 4;
    int numWarps = (block.x * block.y) / 32;
    size_t sharedMemSize = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
    
    size_t dataSize = N * sizeof(int);
    size_t partialHistSize = gridSize * numBins * sizeof(int);
    size_t finalHistSize = numBins * sizeof(int);
    
    // Allocate and initialize host memory.
    int *h_data = (int*) malloc(dataSize);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory for input data.\n");
        return 1;
    }
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_data[i] = rand() % 1024;  // values in [0, 1023]
    }
    
    // Allocate device memory.
    int *d_data, *d_partialHist, *d_finalHist;
    hipMalloc((void**)&d_data, dataSize);
    hipMalloc((void**)&d_partialHist, partialHistSize);
    hipMalloc((void**)&d_finalHist, finalHistSize);
    
    // Copy input data to device.
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);
    
    // Create CUDA events.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    
    // Launch the optimized histogram kernel.
    histogram_optimized_kernel<<<grid, block, sharedMemSize>>>(d_data, d_partialHist, N, numBins);
    
    // Launch the reduction kernel.
    int reduceBlockSize = 256;
    int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
    histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    // Calculate measured throughput based on approximate atomic operations.
    double totalOps = (double) N + (gridSize * numBins); // approximate total operations
    double elapsedSec = elapsedTime / 1000.0;
    double opsPerSec = totalOps / elapsedSec;
    double measuredGFlops = opsPerSec / 1e9;
    
    printf("Total kernel execution time: %f ms\n", elapsedTime);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSec);
    printf("Measured Performance: %f GFLOPS (atomic ops metric)\n", measuredGFlops);
    
    // Calculate occupancy.
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerSM = deviceProp.maxThreadsPerMultiProcessor;
    int activeBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeBlocks, histogram_optimized_kernel, blockSizeTotal, sharedMemSize);
    float occupancy = (activeBlocks * blockSizeTotal) / (float) maxThreadsPerSM;
    occupancy = occupancy * 100.0f;
    printf("Occupancy per SM: %f %%\n", occupancy);
    
    // Display device properties.
    int coresPerSM = 64;
    int totalCores = deviceProp.multiProcessorCount * coresPerSM;
    double clockHz = deviceProp.clockRate * 1000.0;
    double theoreticalOps = totalCores * clockHz * 2;
    printf("Device: %s\n", deviceProp.name);
    printf("Number of SMs: %d\n", deviceProp.multiProcessorCount);
    printf("Cores per SM: %d\n", coresPerSM);
    printf("Total CUDA Cores: %d\n", totalCores);
    printf("Clock Rate: %0.2f GHz\n", clockHz / 1e9);
    printf("Theoretical Peak Ops/sec (int): %e ops/sec\n", theoreticalOps);
    
    // (Optional) Copy final histogram from device to host and print nonzero bins.
    int *h_finalHist = (int*) malloc(finalHistSize);
    hipMemcpy(h_finalHist, d_finalHist, finalHistSize, hipMemcpyDeviceToHost);
    for (int i = 0; i < numBins; i++) {
        if (h_finalHist[i] != 0)
            printf("Bin %d: %d\n", i, h_finalHist[i]);
    }
    
    // Clean up.
    free(h_data);
    free(h_finalHist);
    hipFree(d_data);
    hipFree(d_partialHist);
    hipFree(d_finalHist);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
