#include "hip/hip_runtime.h"
// histogram_atomic.cu
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

namespace cg = cooperative_groups;

// Error-checking macro
#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if(err != hipSuccess) {                                              \
            fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__,   \
                    hipGetErrorString(err));                                 \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while(0)

// Constants: Adjust blockDim and tile size as needed.
#define BLOCK_SIZE 256
// TILE_SIZE defines how many elements each block processes per iteration.
// Here we use double buffering so TILE_SIZE can be e.g. 2 * BLOCK_SIZE.
#define TILE_SIZE (BLOCK_SIZE * 2)

// Kernel: Each block builds a shared-memory histogram using double buffering
// and then atomically adds its result to the global histogram.
__global__ void histogram_kernel(const int *input, int *global_hist,
                                 int num_elements, int num_bins)
{
    // Cooperative groups for warp tiling.
    cg::thread_block cta = cg::this_thread_block();
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    // Shared memory layout:
    // [0, num_bins)         -> shared histogram bins.
    // [num_bins, num_bins + TILE_SIZE)  -> buffer A for input tile.
    // [num_bins + TILE_SIZE, num_bins + 2*TILE_SIZE)  -> buffer B for input tile.
    extern __shared__ int shmem[];
    int *sh_hist = shmem; 
    int *bufferA = shmem + num_bins;
    int *bufferB = bufferA + TILE_SIZE;

    // Initialize shared histogram bins to 0.
    for (int i = threadIdx.x; i < num_bins; i += blockDim.x) {
        sh_hist[i] = 0;
    }
    __syncthreads();

    // Setup double buffering: current buffer pointer and next buffer pointer.
    int *currBuffer = bufferA;
    int *nextBuffer = bufferB;

    // Global offset for this block’s processing.
    // Each block processes TILE_SIZE elements per iteration.
    int base_offset = blockIdx.x * TILE_SIZE;
    int elements_in_tile = 0;

    // Load first tile into currBuffer.
    if (base_offset < num_elements) {
        elements_in_tile = (num_elements - base_offset < TILE_SIZE) ?
                           (num_elements - base_offset) : TILE_SIZE;
        // Each thread loads as many as necessary.
        for (int i = threadIdx.x; i < elements_in_tile; i += blockDim.x) {
            currBuffer[i] = input[base_offset + i];
        }
    }
    __syncthreads();

    // Process tiles until all input elements are handled.
    while (base_offset < num_elements) {
        // Preload next tile into nextBuffer if available.
        int next_base = base_offset + TILE_SIZE;
        int next_elements = 0;
        bool hasNextTile = (next_base < num_elements);
        if (hasNextTile) {
            next_elements = ((num_elements - next_base) < TILE_SIZE) ?
                             (num_elements - next_base) : TILE_SIZE;
            for (int i = threadIdx.x; i < next_elements; i += blockDim.x) {
                nextBuffer[i] = input[next_base + i];
            }
        }
        __syncthreads();

        // Process the current tile in currBuffer.
        // Each thread processes elements in a strided loop.
        for (int i = threadIdx.x; i < elements_in_tile; i += blockDim.x) {
            int val = currBuffer[i];
            // Map value (0-1023) to a bin index.
            // For example, if num_bins==16 then each bin covers 1024/16 = 64 values.
            int bin = val / (1024 / num_bins);
            // Use atomic add in shared memory.
            atomicAdd(&sh_hist[bin], 1);
        }
        __syncthreads();

        // Double buffering swap: move nextBuffer to current.
        currBuffer = nextBuffer;
        elements_in_tile = next_elements;
        base_offset += TILE_SIZE;
        // Synchronize before processing next tile.
        __syncthreads();
    }

    // Finally, each block atomically adds its shared histogram to the global histogram.
    for (int i = threadIdx.x; i < num_bins; i += blockDim.x) {
        atomicAdd(&global_hist[i], sh_hist[i]);
    }
}

// Host function to run the histogram.
int main(int argc, char *argv[])
{
    if(argc != 4 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        printf("Usage: %s -i <BinNum> <VecDim>\n", argv[0]);
        return EXIT_FAILURE;
    }
    int num_bins = atoi(argv[2]);
    int vec_dim = atoi(argv[3]);

    // Validate that num_bins is a power of 2 between 2 and 256.
    bool valid_bins = false;
    for (int k = 2; k <= 8; k++) {
        if(num_bins == (1 << k)) {
            valid_bins = true;
            break;
        }
    }
    if(!valid_bins) {
        fprintf(stderr, "Error: BinNum must be 2^(k) with k between 2 and 8.\n");
        return EXIT_FAILURE;
    }

    // Allocate and initialize host input vector.
    int *h_input = (int *)malloc(vec_dim * sizeof(int));
    if(!h_input) {
        fprintf(stderr, "Host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    srand(time(NULL));
    for (int i = 0; i < vec_dim; i++) {
        // Random integers between 0 and 1023.
        h_input[i] = rand() % 1024;
    }

    // Allocate host histogram output.
    int *h_hist = (int *)malloc(num_bins * sizeof(int));
    if(!h_hist) {
        fprintf(stderr, "Host histogram allocation failed\n");
        free(h_input);
        return EXIT_FAILURE;
    }
    // Initialize histogram to zero.
    for (int i = 0; i < num_bins; i++)
        h_hist[i] = 0;

    // Allocate device memory.
    int *d_input = nullptr;
    int *d_hist = nullptr;
    CUDA_CHECK(hipMalloc((void **)&d_input, vec_dim * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_hist, num_bins * sizeof(int)));

    // Copy host input vector to device.
    CUDA_CHECK(hipMemcpy(d_input, h_input, vec_dim * sizeof(int), hipMemcpyHostToDevice));
    // Initialize device histogram to zero.
    CUDA_CHECK(hipMemset(d_hist, 0, num_bins * sizeof(int)));

    // Determine grid dimensions.
    // Each block processes TILE_SIZE elements.
    int grid_size = (vec_dim + TILE_SIZE - 1) / TILE_SIZE;

    // Calculate shared memory size:
    // shared histogram: num_bins integers +
    // two input buffers: 2*TILE_SIZE integers.
    size_t sharedMemSize = num_bins * sizeof(int) + 2 * TILE_SIZE * sizeof(int);

    // Launch the kernel.
    histogram_kernel<<<grid_size, BLOCK_SIZE, sharedMemSize>>>(d_input, d_hist,
                                                                 vec_dim, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results from device to host.
    CUDA_CHECK(hipMemcpy(h_hist, d_hist, num_bins * sizeof(int), hipMemcpyDeviceToHost));

    // Optionally print the histogram.
    printf("Histogram with %d bins:\n", num_bins);
    for (int i = 0; i < num_bins; i++) {
        printf("Bin %d: %d\n", i, h_hist[i]);
    }

    // Cleanup.
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_hist));
    free(h_input);
    free(h_hist);

    return EXIT_SUCCESS;
}
