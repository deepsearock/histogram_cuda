#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

__global__ void histogram_optimized_kernel(const int *data, int *partialHist, int N, int numBins) {
    extern __shared__ int sharedMem[];

    const int warpSize = 32;
    int blockThreads = blockDim.x * blockDim.y;
    int tileSizeInts = blockThreads * 4;
    int *tile0 = sharedMem;
    int *tile1 = sharedMem + tileSizeInts;
    int numWarps = blockThreads / warpSize;
    int *warpHist = (int*)(sharedMem + 2 * tileSizeInts);

    int k = 0;
    int temp = numBins;
    while (temp > 1) {
        k++;
        temp >>= 1;
    }
    int shift = 10 - k;

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int warp_id = tid / warpSize;
    int lane = tid % warpSize;

    for (int i = lane; i < numWarps * numBins; i += warpSize) {
        warpHist[i] = 0;
    }
    __syncthreads();

    int globalTileSizeInts = gridDim.x * tileSizeInts;
    int firstOffset = blockIdx.x * tileSizeInts;

    if (firstOffset < N) {
        int globalIndex = firstOffset + tid * 4;
        if (globalIndex + 3 < N) {
            int4 tmp = ((const int4*)data)[globalIndex / 4];
            tile0[tid * 4 + 0] = tmp.x;
            tile0[tid * 4 + 1] = tmp.y;
            tile0[tid * 4 + 2] = tmp.z;
            tile0[tid * 4 + 3] = tmp.w;
        } else {
            for (int i = 0; i < 4; i++) {
                int idx = globalIndex + i;
                tile0[tid * 4 + i] = (idx < N) ? data[idx] : -1;
            }
        }
    }
    __syncthreads();

    for (int offset = firstOffset + globalTileSizeInts; offset < N; offset += globalTileSizeInts) {
        int globalIndex = offset + tid * 4;
        if (globalIndex + 3 < N) {
            int4 tmp = ((const int4*)data)[globalIndex / 4];
            tile1[tid * 4 + 0] = tmp.x;
            tile1[tid * 4 + 1] = tmp.y;
            tile1[tid * 4 + 2] = tmp.z;
            tile1[tid * 4 + 3] = tmp.w;
        } else {
            for (int i = 0; i < 4; i++) {
                int idx = globalIndex + i;
                tile1[tid * 4 + i] = (idx < N) ? data[idx] : -1;
            }
        }
        __syncthreads();

        {
            int localBin = -1;
            int localCount = 0;
            #pragma unroll
            for (int i = tid; i < tileSizeInts; i += blockThreads) {
                int value = tile0[i];
                if (value < 0) continue;
                int bin = value >> shift;
                if (bin == localBin) {
                    localCount++;
                } else {
                    if (localCount > 0)
                        atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
                    localBin = bin;
                    localCount = 1;
                }
            }
            if (localCount > 0)
                atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
        }
        __syncthreads();

        int *tempPtr = tile0;
        tile0 = tile1;
        tile1 = tempPtr;
        __syncthreads();
    }

    {
        int localBin = -1;
        int localCount = 0;
        #pragma unroll
        for (int i = tid; i < tileSizeInts; i += blockThreads) {
            int value = tile0[i];
            if (value < 0) continue;
            int bin = value >> shift;
            if (bin == localBin) {
                localCount++;
            } else {
                if (localCount > 0)
                    atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
                localBin = bin;
                localCount = 1;
            }
        }
        if (localCount > 0)
            atomicAdd(&warpHist[warp_id * numBins + localBin], localCount);
    }
    __syncthreads();

    if (warp_id == 0) {
        for (int i = lane; i < numBins; i += warpSize) {
            int sum = 0;
            for (int w = 0; w < numWarps; w++) {
                sum += warpHist[w * numBins + i];
            }
            partialHist[blockIdx.x * numBins + i] = sum;
        }
    }
}

__global__ void histogram_reduce_kernel(const int *partialHist, int *finalHist, int numBins, int numBlocks) {
    int bin = blockIdx.x * blockDim.x + threadIdx.x;
    if (bin < numBins) {
        int sum = 0;
        for (int b = 0; b < numBlocks; b++) {
            sum += partialHist[b * numBins + bin];
        }
        finalHist[bin] = sum;
    }
}

int main(int argc, char *argv[]) {
    if (argc < 4 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        fprintf(stderr, "Usage: %s -i <BinNum> <VecDim> [GridSize]\n", argv[0]);
        return 1;
    }
    
    int numBins = atoi(argv[2]);
    int N = atoi(argv[3]);
    
    if (numBins < 4 || numBins > 256 || (numBins & (numBins - 1)) != 0) {
        fprintf(stderr, "Error: <BinNum> must be 2^k with k from 2 to 8 (e.g., 4, 8, 16, 32, 64, 128, or 256).\n");
        return 1;
    }
    
    const int blockSizeTotal = 64 * 8;
    int gridSize;
    if (argc >= 5)
        gridSize = atoi(argv[4]);
    else
        gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;
    
    dim3 block(64, 8);
    dim3 grid(gridSize);
    
    int tileSizeInts = block.x * block.y * 4;
    int numWarps = (block.x * block.y) / 32;
    size_t sharedMemSize = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
    
    size_t dataSize = N * sizeof(int);
    size_t partialHistSize = gridSize * numBins * sizeof(int);
    size_t finalHistSize = numBins * sizeof(int);
    
    int *h_data = (int*) malloc(dataSize);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory for input data.\n");
        return 1;
    }
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_data[i] = rand() % 1024;
    }
    
    int *d_data, *d_partialHist, *d_finalHist;
    hipMalloc((void**)&d_data, dataSize);
    hipMalloc((void**)&d_partialHist, partialHistSize);
    hipMalloc((void**)&d_finalHist, finalHistSize);
    
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float bestTime = FLT_MAX;
    dim3 bestBlock;
    dim3 bestGrid;
    
    for (int blockX = 32; blockX <= 128; blockX *= 2) {
        for (int blockY = 1; blockY <= 8; blockY *= 2) {
            dim3 block(blockX, blockY);
            int blockSizeTotal = blockX * blockY;
            int gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;
            dim3 grid(gridSize);
            
            int tileSizeInts = block.x * block.y * 4;
            int numWarps = (block.x * block.y) / 32;
            size_t sharedMemSize = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
            
            hipEventRecord(start, 0);
            
            histogram_optimized_kernel<<<grid, block, sharedMemSize>>>(d_data, d_partialHist, N, numBins);
            
            int reduceBlockSize = 256;
            int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
            histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
            
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            
            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);
            
            if (elapsedTime < bestTime) {
                bestTime = elapsedTime;
                bestBlock = block;
                bestGrid = grid;
            }
        }
    }
    
    printf("Best configuration: Block(%d, %d), Grid(%d, %d)\n", bestBlock.x, bestBlock.y, bestGrid.x, bestGrid.y);
    printf("Best execution time: %f ms\n", bestTime);
    
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);
    
    hipEventRecord(start, 0);
    
    histogram_optimized_kernel<<<bestGrid, bestBlock, sharedMemSize>>>(d_data, d_partialHist, N, numBins);
    
    int reduceBlockSize = 256;
    int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
    histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    double totalOps = (double) N + (gridSize * numBins);
    double elapsedSec = elapsedTime / 1000.0;
    double opsPerSec = totalOps / elapsedSec;
    double measuredGFlops = opsPerSec / 1e9;
    
    printf("Total kernel execution time: %f ms\n", elapsedTime);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSec);
    printf("Measured Performance: %f GFLOPS (atomic ops metric)\n", measuredGFlops);
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerSM = deviceProp.maxThreadsPerMultiProcessor;
    int activeBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeBlocks, histogram_optimized_kernel, blockSizeTotal, sharedMemSize);
    float occupancy = (activeBlocks * blockSizeTotal) / (float) maxThreadsPerSM;
    occupancy = occupancy * 100.0f;
    printf("Occupancy per SM: %f %%\n", occupancy);
    
    int coresPerSM = 64;
    int totalCores = deviceProp.multiProcessorCount * coresPerSM;
    double clockHz = deviceProp.clockRate * 1000.0;
    double theoreticalOps = totalCores * clockHz * 2;
    printf("Device: %s\n", deviceProp.name);
    printf("Number of SMs: %d\n", deviceProp.multiProcessorCount);
    printf("Cores per SM: %d\n", coresPerSM);
    printf("Total CUDA Cores: %d\n", totalCores);
    printf("Clock Rate: %0.2f GHz\n", clockHz / 1e9);
    printf("Theoretical Peak Ops/sec (int): %e ops/sec\n", theoreticalOps);
    
    int *h_finalHist = (int*) malloc(finalHistSize);
    hipMemcpy(h_finalHist, d_finalHist, finalHistSize, hipMemcpyDeviceToHost);
    for (int i = 0; i < numBins; i++) {
        if (h_finalHist[i] != 0)
            printf("Bin %d: %d\n", i, h_finalHist[i]);
    }
    
    free(h_data);
    free(h_finalHist);
    hipFree(d_data);
    hipFree(d_partialHist);
    hipFree(d_finalHist);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
