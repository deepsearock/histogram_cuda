#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

// Optimized histogram kernel with double buffering (for input tiling),
// vectorized (int4) loads, and per-warp histograms.
#include <cstdio>
#include <hip/hip_runtime.h>

// First kernel: each block computes a partial histogram in shared memory,
// then writes it to global memory.
__global__ void histogram_optimized_kernel(const int *data,
                                           int *partialHist,
                                           int N,
                                           int numBins)
{
    // Shared memory for one partial histogram per block:
    extern __shared__ int s_Hist[];

    // Zero the shared histogram:
    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        s_Hist[i] = 0;
    }
    __syncthreads();

    // Compute a global thread ID and stride:
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Each thread processes multiple elements in a strided loop:
    while (globalThreadId < N) {
        int value = data[globalThreadId];

        // Compute the bin. (Assume data ranges from 0..1023 for example.)
        // For a general range [0..rangeSize), you'd do bin = value / (rangeSize / numBins).
        // Or adjust logic as needed for your use-case.
        int bin = value / (1024 / numBins);  
        atomicAdd(&s_Hist[bin], 1);

        globalThreadId += stride;
    }

    // Finish all atomic adds to shared memory:
    __syncthreads();

    // Now each block writes its partial histogram into global memory:
    // partialHist has size gridDim.x * numBins
    int offset = blockIdx.x * numBins;
    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        partialHist[offset + i] = s_Hist[i];
    }
}

// Second kernel: sum up the partial histograms from each block into the final histogram.
__global__ void histogram_reduce_kernel(const int *partialHist,
                                        int *finalHist,
                                        int numBins,
                                        int numBlocks)
{
    int bin = blockIdx.x * blockDim.x + threadIdx.x;
    if (bin < numBins) {
        int sum = 0;
        // Accumulate over all blocks:
        for (int b = 0; b < numBlocks; b++) {
            sum += partialHist[b * numBins + bin];
        }
        finalHist[bin] = sum;
    }
}


int main(int argc, char *argv[]) {
    // Usage: ./histogram_atomic -i <BinNum> <VecDim> [BlockSize] [GridSize]
    if (argc < 4 || (argv[1][0] != '-' || argv[1][1] != 'i')) {
        fprintf(stderr, "Usage: %s -i <BinNum> <VecDim> [BlockSize] [GridSize]\n", argv[0]);
        return 1;
    }
    
    int numBins = atoi(argv[2]);
    int N = atoi(argv[3]);
    
    // Validate numBins: must be 2^k with k between 2 and 8.
    if (numBins < 4 || numBins > 256 || (numBins & (numBins - 1)) != 0) {
        fprintf(stderr, "Error: <BinNum> must be 2^k with k from 2 to 8 (e.g., 4, 8, 16, 32, 64, 128, or 256).\n");
        return 1;
    }
    
    // Optionally accept block and grid sizes.
    int blockSizeTotal = 256; // default total threads per block
    int gridSize;
    if (argc >= 5)
        blockSizeTotal = atoi(argv[4]);
    if (argc >= 6)
        gridSize = atoi(argv[5]);
    else
        gridSize = (N + blockSizeTotal - 1) / blockSizeTotal;
    
    // Set 2D block shape: force blockDim.x = 32, blockDim.y = blockSizeTotal / 32.
    int blockDimX = 32;
    int blockDimY = blockSizeTotal / 32;
    if (blockDimY < 1) blockDimY = 1;
    dim3 block(blockDimX, blockDimY);
    dim3 grid(gridSize);
    
    // Calculate shared memory size:
    //   Two tile buffers: 2 * tileSizeInts, where tileSizeInts = block.x * block.y * 4.
    //   Plus per-warp histogram: numWarps * numBins integers.
    int tileSizeInts = block.x * block.y * 4;
    int numWarps = (block.x * block.y) / 32;
    size_t sharedMemSize = (2 * tileSizeInts + numWarps * numBins) * sizeof(int);
    
    size_t dataSize = N * sizeof(int);
    size_t partialHistSize = gridSize * numBins * sizeof(int);
    size_t finalHistSize = numBins * sizeof(int);
    
    // Allocate and initialize host memory.
    int *h_data = (int*) malloc(dataSize);
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory for input data.\n");
        return 1;
    }
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_data[i] = rand() % 1024;  // values in [0, 1023]
    }
    
    // Allocate device memory.
    int *d_data, *d_partialHist, *d_finalHist;
    hipMalloc((void**)&d_data, dataSize);
    hipMalloc((void**)&d_partialHist, partialHistSize);
    hipMalloc((void**)&d_finalHist, finalHistSize);
    
    // Copy input data to device.
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    hipMemset(d_partialHist, 0, partialHistSize);
    hipMemset(d_finalHist, 0, finalHistSize);
    
    // Create CUDA events.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    
    // Launch the optimized histogram kernel.
    histogram_optimized_kernel<<<grid, block, sharedMemSize>>>(d_data, d_partialHist, N, numBins);
    
    // Launch the reduction kernel.
    int reduceBlockSize = 256;
    int reduceGridSize = (numBins + reduceBlockSize - 1) / reduceBlockSize;
    histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize>>>(d_partialHist, d_finalHist, numBins, gridSize);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    // Calculate measured throughput based on approximate atomic operations.
    double totalOps = (double) N + (gridSize * numBins); // approximate total operations
    double elapsedSec = elapsedTime / 1000.0;
    double opsPerSec = totalOps / elapsedSec;
    double measuredGFlops = opsPerSec / 1e9;
    
    printf("Total kernel execution time: %f ms\n", elapsedTime);
    printf("Total operations (approx.): %.0f\n", totalOps);
    printf("Measured Throughput: %e ops/sec\n", opsPerSec);
    printf("Measured Performance: %f GFLOPS (atomic ops metric)\n", measuredGFlops);
    
    // Calculate occupancy.
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerSM = deviceProp.maxThreadsPerMultiProcessor;  // e.g., 2048 for V100
    int activeBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeBlocks, histogram_optimized_kernel, blockSizeTotal, sharedMemSize);
    float occupancy = (activeBlocks * blockSizeTotal) / (float) maxThreadsPerSM;
    occupancy = occupancy * 100.0f;  // percentage
    printf("Occupancy per SM: %f %%\n", occupancy);
    
    // Acquire GPU specifications and calculate theoretical peak integer operations per second.
    // For a Volta-like GPU, assume 64 integer (or FP32) cores per SM and 2 ops per cycle.
    int coresPerSM = 64;
    int totalCores = deviceProp.multiProcessorCount * coresPerSM;
    // deviceProp.clockRate is in kHz; convert to Hz:
    double clockHz = deviceProp.clockRate * 1000.0;
    double theoreticalOps = totalCores * clockHz * 2;  // 2 ops per cycle
    printf("Device: %s\n", deviceProp.name);
    printf("Number of SMs: %d\n", deviceProp.multiProcessorCount);
    printf("Cores per SM: %d\n", coresPerSM);
    printf("Total CUDA Cores: %d\n", totalCores);
    printf("Clock Rate: %0.2f GHz\n", clockHz / 1e9);
    printf("Theoretical Peak Ops/sec (int): %e ops/sec\n", theoreticalOps);
    
    // (Optional) Copy final histogram from device to host and print nonzero bins.
    int *h_finalHist = (int*) malloc(finalHistSize);
    hipMemcpy(h_finalHist, d_finalHist, finalHistSize, hipMemcpyDeviceToHost);
    for (int i = 0; i < numBins; i++) {
        if (h_finalHist[i] != 0)
            printf("Bin %d: %d\n", i, h_finalHist[i]);
    }
    
    // Clean up.
    free(h_data);
    free(h_finalHist);
    hipFree(d_data);
    hipFree(d_partialHist);
    hipFree(d_finalHist);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
